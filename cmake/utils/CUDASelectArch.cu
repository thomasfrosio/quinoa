
#include <hip/hip_runtime.h>
#include <iostream>

int main(int, char**) {
    int count = 0;
    if (hipSuccess != hipGetDeviceCount(&count))
        return -1;
    if (count == 0)
        return -1;
    for (int device = 0; device < count; ++device) {
        if (device)
            std::cout << ';';
        hipDeviceProp_t prop{};
        if (hipSuccess == hipGetDeviceProperties(&prop, device))
            std::cout << prop.major << prop.minor;
        else
            return -1;
    }
    return 0;
}
